/*
* Author Oleksandr Borysov
* Task1
*/


#include <stdio.h>

#include "hip/hip_runtime.h"
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define SEED 34237

#define MAX 32767

__global__ void getPI(double* result, unsigned long* steps) {
	double x, y, z;
	unsigned long count = 0;

	hiprandState_t state;
	hiprand_init(SEED, 0, 0, &state);

	for (unsigned long i = 0; i < *steps; ++i) {
		x = ((double) ((hiprand(&state)) % MAX)) / MAX;
		y = ((double) ((hiprand(&state)) % MAX)) / MAX;
		z = sqrt((x * x) + (y * y));
		if (z <= 1) {
			++count;
		}
	}
	*result = ((double) count / *steps) * 4.0;
}

int main()
{
	double* d_result;
	double result;
	unsigned long* d_stepNumber;
	unsigned long stepNumber;
	double *count_d;
	double count;
	printf("Type number of steps \n");
	scanf("%d", &stepNumber);

	hipMalloc(&d_result, sizeof(double));
	hipMalloc(&d_stepNumber, sizeof(long));

	hipMemcpy(d_result, &result, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_stepNumber, &stepNumber, sizeof(long), hipMemcpyHostToDevice);

	getPI<<<1, 1>>>(d_result, d_stepNumber);

	hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost);

	printf("Calculated PI is = %f.\n", result);
	hipFree(d_result);
	hipFree(d_stepNumber);
    return 0;
}