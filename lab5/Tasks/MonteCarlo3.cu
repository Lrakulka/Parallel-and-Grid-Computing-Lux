/*
* Author Oleksandr Borysov
* Task3
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <ctime>

#define MAX 32767
#define PLOT_DATA_FILE "plot_data3.txt"
#define SEED 254321

__global__ void getCounts(double* results, unsigned long* idxSteps, unsigned long* steps) {
	double x, y, z;
	unsigned long count = 0;
	// index of thread
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t state;
	hiprand_init(SEED + idx, 0, 0, &state);

	for (unsigned long i = 0; i < idxSteps[idx]; ++i) {
		x = ((double)((hiprand(&state)) % MAX)) / MAX;
		y = ((double)((hiprand(&state)) % MAX)) / MAX;
		z = sqrt((x * x) + (y * y));
		if (z <= 1) {
			++count;
		}
	}
	results[idx] = ((double)count / *steps) * 4.0;
}

int main(int argc, char* argv[]) {
	unsigned long stepNumber, threadBlock, threads, threadSteps, threadNumber;
	unsigned long *steps;
	double *results;
	hipError_t cudaStatus;
	clock_t begin = 0;
	
	printf("Type number of steps \n");
	scanf("%lu", &stepNumber);  
	printf("Thread blocks \n");
	scanf("%lu", &threadBlock);  
	printf("Threads in block \n");
	scanf("%lu", &threads); 

	// stepNumber = 1000000; threadBlock = 10; threads = 10;

	begin = clock();
	threadNumber = threadBlock * threads;
	threadSteps = stepNumber / threadNumber;
	results = (double*) calloc(threadNumber, sizeof(double));
	steps = (unsigned long*) calloc(threadNumber, sizeof(long));
	for (int i = 0; i < threadNumber - 1; ++i) {
		steps[i] = threadSteps;
	}
	steps[threadNumber - 1] = stepNumber - threadSteps * (threadNumber - 1);

	unsigned long *d_steps, *d_stepsNumber;
	double *d_results;

	//----------------
	cudaStatus = hipMalloc(&d_results, sizeof(double) * threadNumber);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_results failed!");
        goto Error;
    }
	cudaStatus = hipMalloc(&d_steps, sizeof(long) * threadNumber);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_steps failed!");
        goto Error;
    }
	cudaStatus = hipMalloc(&d_stepsNumber, sizeof(long));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_stepsNumber failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(d_steps, steps, sizeof(long) * threadNumber, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy d_steps failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(d_stepsNumber, &stepNumber, sizeof(long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy d_stepsNumber failed!");
        goto Error;
    }
	// run CUDA method
	getCounts <<<threadBlock, threads>>>(d_results, d_steps, d_stepsNumber);

	cudaStatus = hipMemcpy(results, d_results, sizeof(double) * threadNumber, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy d_result failed! code %d", cudaStatus);
        goto Error;
    }


Error:
	hipFree(d_results);
	hipFree(d_steps);
	hipFree(d_stepsNumber);

	if (cudaStatus == 0) {
		double time_spent = (double) (clock() - begin) / CLOCKS_PER_SEC;
		double pi = 0;
		for (unsigned long i = 0; i < threadNumber; ++i) {
			pi += results[i];
		}
		printf("Calculated PI is = %f.\n Time= %f\n", pi, time_spent);
		FILE* dataPlotFile;
		dataPlotFile = fopen(PLOT_DATA_FILE, "a");
		fprintf(dataPlotFile, "%d %f %d\n", threadNumber, time_spent, stepNumber);
		fclose(dataPlotFile);
	}
	free(results);
	free(steps);
	return cudaStatus;
}