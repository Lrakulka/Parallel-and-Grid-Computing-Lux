/*
* Author Oleksandr Borysov
* Task1
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>

#define SEED 34237

#define MAX 32767

#define PLOT_DATA_FILE "plotData_1.txt"

__global__ void getPI(double* result, unsigned long* steps) {
	double x, y, z;
	long count = 0;

	hiprandState_t state;
	hiprand_init(SEED, 0, 0, &state);

	for (unsigned long i = 0; i < *steps; ++i) {
		x = ((double) ((hiprand(&state)) % MAX)) / MAX;
		y = ((double) ((hiprand(&state)) % MAX)) / MAX;
		z = sqrt((x * x) + (y * y));
		if (z <= 1) {
			++count;
		}
	}
	*result = ((double) count / *steps) * 4.0;
}

int main()
{
	hipError_t cudaStatus;
	double* d_result;
	double result;
	unsigned long* d_stepNumber;
	unsigned long stepNumber;
	clock_t begin = clock();

	printf("Type number of steps \n");
	scanf("%d", &stepNumber);
	// Allocate memory in GPU
	cudaStatus = hipMalloc(&d_result, sizeof(double));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_result failed!");
        goto Error;
    }
	cudaStatus = hipMalloc(&d_stepNumber, sizeof(long));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_stepNumber failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(d_stepNumber, &stepNumber, sizeof(long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy d_stepNumber failed!");
        goto Error;
    }
	// Call cuda method
	getPI<<<1, 1>>>(d_result, d_stepNumber);

	 // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	// Chek result
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "getPI failed!");
        return 1;
    }

	cudaStatus = hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy result failed!");
        goto Error;
    }
	printf("Calculated PI is = %f.\n", result);

Error:
	hipFree(d_result);
	hipFree(d_stepNumber);
    
	if (cudaStatus == 0) {
		double time_spent = (double) (clock() - begin) / CLOCKS_PER_SEC;
		FILE* dataPlotFile;
		dataPlotFile = fopen(PLOT_DATA_FILE, "a");
		fprintf(dataPlotFile, "%d %f\n", stepNumber, time_spent);
		fclose(dataPlotFile);
		printf("%d %f\n", stepNumber, time_spent);
	}
    return cudaStatus;
}
	
